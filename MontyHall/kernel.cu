
#include "hip/hip_runtime.h"

#include "hiprand/hiprand.h"
#include "hiprand/hiprand_kernel.h"

#include <stdio.h>
#include <time.h>

__global__ void mhNoSwitch(unsigned int *randNums, bool *results)
{
	unsigned int idx = blockIdx.x * 1024 + threadIdx.x;
	unsigned int rand0 = randNums[2*idx];
	unsigned int rand1 = randNums[2*idx+1];
	// door with prize
	unsigned int prize = rand0 % 3;
	// door choice
	unsigned int choice = rand1 % 3;
	// since choice does not change after extra door opening, just check if choice and prize door are the same
	results[idx] = (prize == choice);
}

__global__ void mhSwitch(unsigned int *randNums, bool *results)
{
	unsigned int idx = blockIdx.x * 1024 + threadIdx.x;
	unsigned int rand0 = randNums[2*idx];
	unsigned int rand1 = randNums[2*idx+1];

	// door with prize
	unsigned int prize = rand0 % 3;
	// initial choice
	unsigned int choice = rand1 % 3;

	unsigned int reveal = 0;
	unsigned int final = 0;
	if (prize == 0) {
		if (choice == 0) {
			reveal = 1 + rand0 % 2;
			final = 3 - reveal;
		} else if (choice == 1) {
			reveal = 2;
			final = 0;
		} else { // choice == 2
			reveal = 1;
			final = 0;
		}
	} else if (prize == 1) {
		if (choice == 0) {
			reveal = 2;
			final = 1;
		} else if (choice == 1) {
			reveal = 2 * (rand0 % 2);
			final = 2 - reveal;
		} else { // choice == 2
			reveal = 0;
			final = 1;
		}
	} else if (prize == 2) {
		if (choice == 0) {
			reveal = 1;
			final = 2;
		} else if (choice == 1) {
			reveal = 0;
			final = 2;
		} else { // choice == 2
			reveal = rand0 % 2;
			final = 1 - reveal;
		}
	}

	results[idx] = (prize == final);
}

int main()
{
	hipSetDevice(0);

	const unsigned int numBlocks = 10;
	const unsigned int numTests = 1024 * numBlocks;
	bool results_noswitch[numTests];
	bool results_switch[numTests];
	bool *dev_results_noswitch = 0;
	bool *dev_results_switch = 0;
	unsigned int *dev_randNums = 0;

	hipMalloc((void**) &dev_results_noswitch, numTests * sizeof(bool));
	hipMalloc((void**) &dev_results_switch, numTests * sizeof(bool));
	hipMalloc((void**) &dev_randNums, 2 * numTests * sizeof(unsigned int));

	hiprandGenerator_t generator;
	hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(generator, time(NULL));
	hiprandGenerate(generator, dev_randNums, 2 * numTests);

	mhNoSwitch<<<numBlocks, 1024>>>(dev_randNums, dev_results_noswitch);
	mhSwitch<<<numBlocks, 1024>>>(dev_randNums, dev_results_switch);

	hipMemcpy(results_noswitch, dev_results_noswitch, numTests * sizeof(bool), hipMemcpyDeviceToHost);
	hipMemcpy(results_switch, dev_results_switch, numTests * sizeof(bool), hipMemcpyDeviceToHost);
	hipFree(dev_results_noswitch);
	hipFree(dev_results_switch);
	hiprandDestroyGenerator(generator);

	unsigned int noswitchpass = 0;
	unsigned int switchpass = 0;
	for (int i = 0; i < numTests; i++) {
		noswitchpass += results_noswitch[i] ? 1 : 0;
		switchpass += results_switch[i] ? 1 : 0;
	}

	printf("no switch picked correctly %i out of %i, which is %g percent\n", noswitchpass, numTests, 100 * ((float) noswitchpass) / ((float) numTests));
	printf("switch picked correctly %i out of %i, which is %g percent\n", switchpass, numTests, 100 * ((float) switchpass) / ((float) numTests));

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
